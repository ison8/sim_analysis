#include "hip/hip_runtime.h"
#include "analysis_header.h"

template<int BLOCK_SIZE> __global__ void calcAve(double *tmp,double *ave,double white_x,double white_y, double array_row, double array_col) {
    /* ���b�V���p�ϐ� */
    double mesh_x_start = (double)blockIdx.x * 0.2;
    double mesh_x_end = (double)blockIdx.x * 0.2 + 0.2;
    double mesh_y_start = (double)threadIdx.x * 0.2;
    double mesh_y_end = (double)threadIdx.x * 0.2 + 0.2;

    /* ���όv�Z */
    double ave_deg = 0;
    double ave_count = 0;


}

int main(void) {
    /* �t�@�C���� */
    string filename = "C:/Users/ryoin/source/repos/ison8/color_simulation_ver1/color_simulation_ver1/sim_result.csv";
    
    /* CSV�t�@�C���̒l���i�[����x�N�^�[ */
    vector<string> sim_data;
    /* �t�@�C���T�C�Y�A�t�@�C���s���A�t�@�C���񐔂��i�[����z�� */
    double f_size[3] = { 0,0,0 };
    
    /* CSV�t�@�C���ǂݍ��� */
    readCSV(sim_data, filename,f_size);

    /* �f�[�^������vector��` */
    vector<double> arranged_data((int)f_size[2]);

    /* �ǂݍ��񂾃f�[�^�𐮓ڂ��� */
    arrangeData(sim_data, arranged_data, f_size);

    /* CUDA�֓n���p�̕ϐ� */
    double* tmp;
    double* ave;

    double white_x = arranged_data[0];
    double white_y = arranged_data[f_size[0]];

    /* �z��̓��I�m�� */
    tmp = new double[f_size[1] * 3 * 50];
    ave = new double[45 * 45 * 2];

    /* tmp�̑傫�� */
    int tmp_size = f_size[0] * 3 * 50;

    /* �u���b�N�T�C�Y�A�O���b�h�T�C�Y */
    int grid = 45, block = 45;

    /* for���ŉ񂵂ĕ��ς��v�Z���Ă��� */
    for (int i = 0; i < ((int)f_size[2]); i += tmp_size) {
 
    }
    makeCpy(arranged_data, tmp, 0,tmp_size,f_size[2]);

    return 0;
}